
#include <hip/hip_runtime.h>
#include <stdlib.h>
#include <stdio.h>

#define SIZE int(pow(2,10))
#define THREADS 1024
#define BLOCKS SIZE/THREADS

__global__ void BitonicSort(int* arr_d);
void CheckSolution(int* arr);
void PrintArray(int* arr);

int main()
{
	int* arr=new int[SIZE];
	for(int i=0;i<SIZE;i++)
	arr[i]=rand()%10;

	int* arr_d;
	hipMalloc((void**)&arr_d,SIZE*sizeof(int));
	hipMemcpy(arr_d,arr,SIZE*sizeof(int),hipMemcpyHostToDevice);
	
	BitonicSort<<<BLOCKS,THREADS>>>(arr_d);

	hipMemcpy(arr,arr_d,SIZE*sizeof(int),hipMemcpyDeviceToHost);
	CheckSolution(arr);
}

__global__ void BitonicSort(int* arr_d)
{
	int i=threadIdx.x;
	__shared__ int Shared_arr[THREADS];
	Shared_arr[i]=arr_d[i];
	
	for(int k=2;k<=THREADS;k<<=1)
	for(int j=k>>1;j>0;j>>=1)
	{
		int ixj=i^j;
		if(i<ixj)
		{	// Sort ascending 
			if((i&k)==0)
			{
				if(Shared_arr[i]>Shared_arr[ixj])
				{
					int temp=Shared_arr[i];
					Shared_arr[i]=Shared_arr[ixj];
					Shared_arr[ixj]=temp;
				}
			}
			// Sort descending 
			else
			{
				if(Shared_arr[i]<Shared_arr[ixj])
				{
					int temp=Shared_arr[i];
					Shared_arr[i]=Shared_arr[ixj];
					Shared_arr[ixj]=temp;
				}
			}
		} 
		
		__syncthreads();
	}
	
	arr_d[i]=Shared_arr[i];
}

void CheckSolution(int* arr)
{
	int i;
	for(i=0;i<SIZE-1;i++)
	if(arr[i]>arr[i+1])
	{
		printf("Solution is Wrong!\n");
		break;
	}
	
	if(i==SIZE-1)
	printf("Solution is right!\n");
}

void PrintArray(int* arr)
{
	for(int i=0;i<SIZE;i++)
	printf("%d ",arr[i]);

	printf("\n");
}