
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<time.h>

#define NUM_BINS 4096
#define BIN_CAP 127

__global__ void Histogram(const int*,int*,int);
bool CheckAnswer(int*,int*,int);
void printArray(int*,int);


int main()
{	
	/* Variables to measure the overhead time*/
	clock_t start_overhead,end_overhead;
	start_overhead=clock();
	
	int nROW=12000;	
	
	int* Array=new int[nROW];
	int* HostBins=new int[NUM_BINS];
	
	srand(time(NULL));
	
	/* Populate the input array*/
	for(int i=0;i<nROW;i++)
	Array[i]=rand()%NUM_BINS;
  
	int ArraySize=nROW*sizeof(int);
	int BinSize=NUM_BINS*sizeof(int);
	
	int* Array_d, *DeviceBins;
	
	/* Allocate memory on the device*/
	
	hipMalloc((void**)&Array_d,ArraySize);
	hipMalloc((void**)&DeviceBins,BinSize);
	
	/* Copy data from host to device*/
	hipMemcpy(Array_d,Array,ArraySize,hipMemcpyHostToDevice);
	hipMemcpy(DeviceBins,HostBins,BinSize,hipMemcpyHostToDevice);

	/* Variables to measure device computation time*/
	clock_t start,end,total;
	start=clock();
	
	/* Kernel launch*/
	Histogram<<<ceil(nROW/1024.0),1024>>>(Array_d,DeviceBins,nROW);
	
	end=clock();
	total=(double)(end - start) / CLOCKS_PER_SEC;
	printf("Time taken on device: %lf\n",total);
	
	/* Copy the histogram to host*/
	hipMemcpy(HostBins,DeviceBins,BinSize,hipMemcpyDeviceToHost);
	
	end_overhead=clock();
	
	/* Verify solution*/
	if(CheckAnswer(Array,HostBins,nROW))
	printf("Solution is right!\n");
    else
	printf("Solution is wrong!\n");
	
	printf("Time spent on overhead calculations: %lf\n",(double)(end_overhead - start_overhead) / CLOCKS_PER_SEC);
	
/* 	printArray(Array,nROW);
	printArray(HostBins,NUM_BINS); */
	
	
	hipFree(Array_d);
	hipFree(DeviceBins);
}

__global__ void Histogram(const int * __restrict__ deviceInput,int *deviceBins,int inputLength)
{
	int row=blockIdx.x*blockDim.x+threadIdx.x;
	deviceBins[row]=0;
	
	if(row<inputLength)
	{
		int element=deviceInput[row];
		if(deviceBins[element]<BIN_CAP)
	    atomicAdd(&deviceBins[element],1);
	}
}

/* A function to verify correctness of solution*/
bool CheckAnswer(int* Array,int* HostBins, int nROW )
{		
	int * temp =new int[NUM_BINS];
	
	clock_t start,end;
	start=clock();
	
	for(int i=0;i<NUM_BINS;i++)
	temp[i]=0;

	for(int i=0;i<nROW;i++)
	{
		if(temp[Array[i]]<BIN_CAP)
		temp[Array[i]]++;
	}
	
	end=clock();
	printf("Time taken on host: %lf\n",(double)(end - start) / CLOCKS_PER_SEC);
	
	for(int i=0;i<NUM_BINS;i++)
	{
		if(temp[i]!=HostBins[i])
		{
			printf("Location->%d,Expected->%d,Received->%d",i,temp[i],HostBins[i]);
			return false;
		}
	}
	return true;
	
}

void printArray(int* arr,int size)
{
	for(int i=0;i<size;i++)
	printf("%d\n",arr[i]);

	printf("\n");
}