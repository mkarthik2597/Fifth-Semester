#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include "wb.h"

int main(int argc, char *argv[]) {
  wbArg_t args;
  float *hostInput;
  float hostOutput;
  int inputLength;

  args = wbArg_read(argc, argv); /* parse the input arguments */

  // Import host input data
  wbTime_start(Generic, "Importing data to host");
  hostInput = (float *)wbImport(wbArg_getInputFile(args, 0), &inputLength);
  wbTime_stop(Generic, "Importing data to host");

  wbTime_start(GPU, "Doing GPU Computation (memory + compute)");
  // Declare and allocate thrust device input and output vectors
  wbTime_start(GPU, "Doing GPU memory allocation");
  //@@ Insert code here
  thrust::device_vector<float> deviceInput(hostInput,hostInput+inputLength);
  wbTime_stop(GPU, "Doing GPU memory allocation");

  // Copy to device
  wbTime_start(Copy, "Copying data to the GPU");
  //@@ Insert code here
  
  wbTime_stop(Copy, "Copying data to the GPU");

  // Execute vector addition
  wbTime_start(Compute, "Doing the computation on the GPU");
  //@@ Insert Code here
  hostOutput=thrust::reduce(deviceInput.begin(), deviceInput.end());
  wbTime_stop(Compute, "Doing the computation on the GPU");
  /////////////////////////////////////////////////////////

  wbTime_stop(GPU, "Doing GPU Computation (memory + compute)");

  wbSolution(args, &hostOutput, 1);

  free(hostInput);
  return 0;
}
