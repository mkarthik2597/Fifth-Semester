
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<math.h>

#define size int(pow(2,25))

__global__ void reduction(int*);
void printArray(int*);

int main()
{
	int* input=new int[size];
	for(int i=0;i<size;i++)
	input[i]=rand()%100;

	int* input_d;

	hipMalloc((void**)&input_d,size*sizeof(int));
	hipMemcpy(input_d,input,size*sizeof(int),hipMemcpyHostToDevice);

	dim3 dimBlock(1024);
	/* Number of blocks are halved with block size being the same*/
	int nBlocks=size/1024/2;
	dim3 dimGrid(ceil(nBlocks));

	int Solution=0;
	for(int i=0;i<size;i++)
	Solution+=input[i];


	reduction<<<dimGrid,dimBlock,dimBlock.x*sizeof(int)>>>(input_d);

	hipMemcpy(input,input_d,size*sizeof(int),hipMemcpyDeviceToHost);


	int Answer=0;
	for(int i=0;i<size;i+=2*dimBlock.x)
	Answer+=input[i];

	if(Solution==Answer)
	printf("Solution is right\n");
	else
	{
		printf("Solution is wrong\n");
		printf("Answer:%d\n",Answer);
	}

}

__global__ void reduction(int* input_d)
{
	int tx=threadIdx.x,bx=blockIdx.x;
	int inx=2*bx*blockDim.x+tx;

	extern __shared__ int partialSum[];
	/* There are no idle threads in this reduction */
	partialSum[tx]=input_d[inx]+input_d[inx+blockDim.x];
	__syncthreads();

	for(int s=blockDim.x/2;s>0;s>>=1)
	{
		if(tx<s)
		partialSum[tx]+=partialSum[tx+s];

		__syncthreads();
	}

	if(tx==0)
	input_d[inx]=partialSum[0];
}

void printArray(int* arr)
{
	for(int i=0;i<size;i++)
	printf("%d ",arr[i]);

	printf("\n");
}
